#include "hip/hip_runtime.h"
#include "qdbmp.h"
#include "stdio.h"
#include "util.cuh"
#include <cuda_device_runtime_api.h>

__device__ int d_abs(int a) { return a > 0 ? a : -a; }


#define cudaCheckForErrorAndSync()                                             \
  gpuErrchk(hipPeekAtLastError());                                            \
  gpuErrchk(hipDeviceSynchronize());

#define cudaStartTimer(start, stop)                                            \
  hipEventCreate(&start);                                                     \
  hipEventCreate(&stop);                                                      \
  hipEventRecord(start, 0);

#define cudaStopTimerAndRecord(start, stop, time)                              \
  hipEventRecord(stop, 0);                                                    \
  hipEventSynchronize(stop);                                                  \
  hipEventElapsedTime(&time, start, stop);

#define span 1
#define divider 4

__device__ void energy_pixel(UCHAR *r, UCHAR *g, UCHAR *b, UCHAR *o_r,
                             UCHAR *o_g, UCHAR *o_b, int width, int height) {

  o_r[0] = (d_abs(r[0] - r[span]) + d_abs(r[0] - r[-span]) +
            d_abs(r[0] - r[width * span]) + d_abs(r[0] - r[-width * span])) /
           // d_abs(r[0] - r[width * span + span]) +
           // d_abs(r[0] - r[width * span - span]) +
           // d_abs(r[0] - r[-width * span + span]) +
           // d_abs(r[0] - r[-width * span - span])) /
           divider;
  o_g[0] = (d_abs(g[0] - g[span]) + d_abs(g[0] - g[-span]) +
            d_abs(g[0] - g[width * span]) + d_abs(g[0] - g[-width * span])) /
           // d_abs(g[0] - g[width * span + span]) +
           // d_abs(g[0] - g[width * span - span]) +
           // d_abs(g[0] - g[-width * span + span]) +
           // d_abs(g[0] - g[-width * span - span])) /
           divider;
  o_b[0] = (d_abs(b[0] - b[span]) + d_abs(b[0] - b[-span]) +
            d_abs(b[0] - b[width * span]) + d_abs(b[0] - b[-width * span])) /
           // d_abs(b[0] - b[width * span + span]) +
           // d_abs(b[0] - b[width * span - span]) +
           // d_abs(b[0] - b[-width * span + span]) +
           // d_abs(b[0] - b[-width * span - span])) /
           divider;
  UCHAR gray = o_r[0] * 0.3 + o_g[0] * 0.59 + o_b[0] * 0.11;
  o_r[0] = gray > 255 ? 255 : gray;
  o_g[0] = gray > 255 ? 255 : gray;
  o_b[0] = gray > 255 ? 255 : gray;
}

__global__ void energy_kernel(UCHAR *r, UCHAR *g, UCHAR *b, UCHAR *o_r,
                              UCHAR *o_g, UCHAR *o_b, int width, int height) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  if (x >= width || y >= height) {
    return;
  }
  int index = x + y * width;
  energy_pixel(&r[index], &g[index], &b[index], &o_r[index], &o_g[index],
               &o_b[index], width, height);
}

extern "C" void energy(UCHAR *out_r, UCHAR *out_g, UCHAR *out_b, int width,
                       int height) {
  dim3 threads(32, 32);
  dim3 blocks(((width - 1) / 32) + 1, ((height - 1) / 32) + 1);
  unsigned int pixelCount = width * height;

  UCHAR *original_r;
  UCHAR *original_g;
  UCHAR *original_b;
  hipMalloc((void **)&original_r, pixelCount * sizeof(UCHAR));
  hipMalloc((void **)&original_g, pixelCount * sizeof(UCHAR));
  hipMalloc((void **)&original_b, pixelCount * sizeof(UCHAR));

  hipMemcpy(original_r, out_r, pixelCount * sizeof(UCHAR),
             hipMemcpyHostToDevice);
  hipMemcpy(original_g, out_g, pixelCount * sizeof(UCHAR),
             hipMemcpyHostToDevice);
  hipMemcpy(original_b, out_b, pixelCount * sizeof(UCHAR),
             hipMemcpyHostToDevice);

  UCHAR *energy_r;
  UCHAR *energy_g;
  UCHAR *energy_b;
  hipMalloc((void **)&energy_r, pixelCount * sizeof(UCHAR));
  hipMalloc((void **)&energy_g, pixelCount * sizeof(UCHAR));
  hipMalloc((void **)&energy_b, pixelCount * sizeof(UCHAR));

    float time;
  hipEvent_t start, stop;
  cudaStartTimer(start, stop);

  energy_kernel<<<blocks, threads>>>(original_r, original_g, original_b,
                                     energy_r, energy_g, energy_b, width,
                                     height);
  cudaCheckForErrorAndSync();
  cudaStopTimerAndRecord(start, stop, time);
  printf("GPU kernel took %.4f ms \n\n", time);

  hipMemcpy(out_r, energy_r, pixelCount * sizeof(UCHAR),
             hipMemcpyDeviceToHost);
  hipMemcpy(out_g, energy_g, pixelCount * sizeof(UCHAR),
             hipMemcpyDeviceToHost);
  hipMemcpy(out_b, energy_b, pixelCount * sizeof(UCHAR),
             hipMemcpyDeviceToHost);

  hipFree(original_r);
  hipFree(original_g);
  hipFree(original_b);
  hipFree(energy_r);
  hipFree(energy_g);
  hipFree(energy_b);

  int desaturation = 1;
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      int index = x + y * width;
      *(out_r + index) = *(out_r + index) / desaturation;
      *(out_g + index) = *(out_g + index) / desaturation;
      *(out_b + index) = *(out_b + index) / desaturation;
    }
  }
}
